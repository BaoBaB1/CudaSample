﻿#include "hip/hip_runtime.h"


#include <vector>
#include <iostream>
#include <numeric>
#include <cassert>
#include <chrono>

#define MEASURE_EXEC_TIME(funccall) \
  auto start = std::chrono::high_resolution_clock::now(); \
  funccall; \
  auto stop = std::chrono::high_resolution_clock::now(); \
  std::cout << std::chrono::duration<double, std::milli>(stop - start).count() << '\n';

template<typename T>
__global__ void add1D(T* a, T* b, T* c, int N)
{
  // 1-dim grid blocks:
  // (0, 0), (1, 0), (2, 0) ...
  // or 
  // (0, 0)
  // (0, 1)
  // (0, 2) 
  // ...

  // 2-dim grid blocks:
  // (0, 0), (1, 0), (2, 0) ...
  // (0, 1), (1, 1), (2, 1) ...
  // (0, 2), (1, 2), (2, 2) ...
  // ...

  int i = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) +
    threadIdx.y * blockDim.x +
    threadIdx.x;
  if (i < N) {
    c[i] = a[i] + b[i];
  }
}

template<typename T>
__global__ void add2D(T* a, T* b, T* c, int rows, int cols)
{
  int i = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) +
    threadIdx.y * blockDim.x +
    threadIdx.x;
  if (i < rows * cols) {
    c[i] = a[i] + b[i];
  }
}

template<typename T>
void add1DOnCpu(T* a, T* b, T* c, int N)
{
  for (int i = 0; i < N; i++) {
    c[i] = a[i] + b[i];
  }
}

template<typename T>
void add2DOnCpu(T* a, T* b, T* c, int width, int height)
{
  for (int i = 0; i < width; i++) {
    for (int j = 0; j < height; j++) {
      c[i * width + j] = a[i * width + j] + b[i * width + j];
    }
  }
}

template<typename T, int N>
void testPerformance(const dim3 gridSize, const dim3 threadsPerBlock, const int arrayDim)
{
  T cpuRes = 0;
  T cudaRes = 0;

  std::cout << "Array dimension == " << arrayDim << ", N elems == " << (arrayDim == 1 ? N : N * N) << '\n';

  if (arrayDim == 1)
  {

    // on cpu
    {
      std::vector<T> a(N), b(N), c(N);
      for (int i = 0; i < N; i++) {
        a[i] = i + 1;
        b[i] = (i + 1) * 2;
      }

      std::cout << "Time taken on cpu: ";
      MEASURE_EXEC_TIME(add1DOnCpu(a.data(), b.data(), c.data(), N));
      cpuRes = std::accumulate(c.begin(), c.end(), 0);
    }

    // on cuda
    {
      std::vector<T> a(N), b(N), c(N);
      const size_t bytes = sizeof(T) * N;
      for (int i = 0; i < N; i++) {
        a[i] = i + 1;
        b[i] = (i + 1) * 2;
      }

      T* da, * db, * dc;
      hipMalloc(&da, bytes);
      hipMalloc(&db, bytes);
      hipMalloc(&dc, bytes);

      hipMemcpy(da, a.data(), bytes, hipMemcpyKind::hipMemcpyHostToDevice);
      hipMemcpy(db, b.data(), bytes, hipMemcpyKind::hipMemcpyHostToDevice);

      std::cout << "Time taken on gpu: ";
      MEASURE_EXEC_TIME((add1D<<<gridSize, threadsPerBlock>>>(da, db, dc, N)));
      std::cout << '\n';

      hipMemcpy(c.data(), dc, bytes, hipMemcpyKind::hipMemcpyDeviceToHost);
      cudaRes = std::accumulate(c.begin(), c.end(), 0);

      hipFree(da);
      hipFree(db);
      hipFree(dc);
    }
  }
  else if (arrayDim == 2)
  {

    // on cpu
    {
      std::vector<T> a(N * N);
      std::vector<T> b(N * N);
      std::vector<T> c(N * N);
      for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
          a[i * N + j] = i * N + j; 
          b[i * N + j] = std::pow(i * N + j, 2);
        }
      }

      std::cout << "Time taken on cpu: ";
      MEASURE_EXEC_TIME(add2DOnCpu(a.data(), b.data(), c.data(), N, N));
      cpuRes = std::accumulate(c.begin(), c.end(), 0);
    }

    // on cuda
    {
      std::vector<T> a(N * N), b(N * N), c(N * N);
      const size_t bytes = sizeof(T) * N * N;
      for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
          a[i * N + j] = i * N + j;
          b[i * N + j] = std::pow(i * N + j, 2);
        }
      }

      T* da, * db, * dc;
      hipMalloc(&da, bytes);
      hipMalloc(&db, bytes);
      hipMalloc(&dc, bytes);

      hipMemcpy(da, a.data(), bytes, hipMemcpyKind::hipMemcpyHostToDevice);
      hipMemcpy(db, b.data(), bytes, hipMemcpyKind::hipMemcpyHostToDevice);

      std::cout << "Time taken on gpu: ";
      MEASURE_EXEC_TIME((add2D<<<gridSize, threadsPerBlock>>>(da, db, dc, N, N)));
      std::cout << '\n';

      hipMemcpy(c.data(), dc, bytes, hipMemcpyKind::hipMemcpyDeviceToHost);
      cudaRes = std::accumulate(c.begin(), c.end(), 0);

      hipFree(da);
      hipFree(db);
      hipFree(dc);
    }
  }
  else
  {
    throw std::runtime_error("Not implemented\n");
  }

  if (cpuRes != cudaRes) {
    std::cout << "Array dim = " << arrayDim << ", N = " << N << ", cpuRes = " 
      << cpuRes << ", cudaRes = " << cudaRes << '\n';
  }
  assert(cpuRes == cudaRes);
}

int main()
{

  {
    // 1 row, 2 cols, 1D array
    dim3 gridSize(2, 1);
    dim3 threadsPerBlock(16, 16);
    testPerformance<int, 512>(gridSize, threadsPerBlock, 1);
  }

  {
    // 4 rows, 1 col, 1D array
    dim3 gridSize(1, 4);
    dim3 threadsPerBlock(16, 16);
    testPerformance<int, 1024>(gridSize, threadsPerBlock, 1);
  }

  {
    // 14 rows, 15 cols, 1D array
    dim3 gridSize(15, 14);
    dim3 threadsPerBlock(16, 16);
    testPerformance<size_t, 50000>(gridSize, threadsPerBlock, 1);
  }

  {
    // 14 rows, 15 cols, 1D array
    dim3 gridSize(30, 30);
    dim3 threadsPerBlock(24, 24);
    testPerformance<size_t, 500000>(gridSize, threadsPerBlock, 1);
  }

  {
    // 1 row, 3 cols, 2D array
    dim3 gridSize(3, 1);
    dim3 threadsPerBlock(12, 12);
    testPerformance<size_t, 16>(gridSize, threadsPerBlock, 2);
  }

  {
    // 22 rows, 22 cols, 2D array
    dim3 gridSize(22, 22);
    dim3 threadsPerBlock(24, 24);
    testPerformance<size_t, 512>(gridSize, threadsPerBlock, 2);
  }
}
